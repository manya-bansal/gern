#include "hip/hip_runtime.h"
// From https:github.com/SzymonOzog/FastSoftmax.git

#include "benchmark.h"
#include "impl/gpu-matrix-const.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<int stride,
         typename T,
         typename T2>
__device__ void query(T a,
                      int x,
                      int y,
                      T2 &reg_array_big) {
    auto &reg_array = reg_array_big.array;
    float4 *val_ptr = reinterpret_cast<float4 *>(&a.data[x * a.row + y * 4]);

    constexpr int64_t num_row = reg_array_big.rows;
    constexpr int64_t num_col = reg_array_big.cols_by_4;

    int index = 0;
    for (int m = 0; m < num_row; m++) {
#pragma unroll URF
        for (int i = 0; i < num_col; i++) {
            float4 val = val_ptr[index];
            reg_array[i] = val;
            index += stride;
        }
    }
}

template<int stride,
         typename T1,
         typename T2>
__device__ void insert(T1 a,
                       int x,
                       int y,
                       T2 &reg_array_big) {
    auto &reg_array = reg_array_big.array;
    constexpr int64_t num_row = reg_array_big.rows;
    constexpr int64_t num_col = reg_array_big.cols_by_4;
    float4 *val_ptr = reinterpret_cast<float4 *>(&a.data[x * a.row + y * 4]);

    int index = 0;
    for (int m = 0; m < num_row; m++) {
#pragma unroll URF
        for (int i = 0; i < num_col; i++) {
            float4 val = reg_array[i];
            val_ptr[index] = val;
            index += stride;
        }
    }
}

template<int num_row, int num_col>
__device__ StaticMatrix<num_row, num_col> allocate_local() {
    return StaticMatrix<num_row, num_col>();
}

#include "impl/impl.h"

template<int tile_row,
         int tile_col,
         typename T>
__global__ void softmax_kernel_mine(T a,
                                    T b) {
    int x = blockIdx.x;
    int y = threadIdx.y;

    constexpr int64_t num_cols_in = CEILING((a.col / 4), tile_col);
    constexpr int64_t num_rows_in = tile_row;

    // if (y < h)
    // {
    StaticMatrix<num_rows_in, num_cols_in> reg_array_big;
    query<tile_col>(a, x, y, reg_array_big);
    holder<num_rows_in> hold;
    max_shuffle<tile_col>(hold, reg_array_big);
    subtract_vec(hold, reg_array_big, reg_array_big);
    exp_matrix(reg_array_big, reg_array_big);
    sum_row<tile_col>(hold, reg_array_big);
    divide_vec(hold, reg_array_big, reg_array_big);
    insert<tile_col>(b, x, y, reg_array_big);
    // }
}

template<int64_t col, int64_t col_val, int64_t row, int64_t stride_val>
__global__ void function_39(impl::MatrixGPU<16384, 16384, 16384, 1024> input,
                            impl::MatrixGPU<16384, 16384, 16384, 1024> output) {

    int64_t _gern_x_3_24_35 = ((blockIdx.x * row) + 0);
    // for (int64_t _gern_y_4_26 = 0; (_gern_y_4_26 < output.col); _gern_y_4_26 = (_gern_y_4_26 + col_val)) {

    int64_t _gern_y_4 = threadIdx.y;
    constexpr int64_t _gern_l_y_2 = col_val;
    int64_t _gern_x_3 = _gern_x_3_24_35;
    constexpr int64_t _gern_l_x_1 = row;

    int64_t _gern_x_6 = _gern_x_3;
    constexpr int64_t _gern_l_x_5 = _gern_l_x_1;
    int64_t _gern_x_9 = _gern_x_3;
    int64_t _gern_y_10 = _gern_y_4;
    constexpr int64_t _gern_l_x_7 = _gern_l_x_1;
    constexpr int64_t _gern_l_y_8 = _gern_l_y_2;
    int64_t _gern_x_13 = _gern_x_9;
    int64_t _gern_y_14 = _gern_y_10;
    constexpr int64_t _gern_l_x_11 = _gern_l_x_7;
    constexpr int64_t _gern_l_y_12 = _gern_l_y_8;
    int64_t _gern_x_16 = _gern_x_13;
    constexpr int64_t _gern_l_x_15 = _gern_l_x_11;

    auto _query_output_40 = output.query_new<_gern_l_x_1, _gern_l_y_2>(_gern_x_3, _gern_y_4);

    auto max_row_out = impl::allocate_static_array<_gern_l_x_15>();

    auto _query_input_41 = input.query_new<_gern_l_x_15, col>(_gern_x_16, 0);

    max_shuffle<stride_val>(max_row_out, _query_input_41);

    auto sub_temp = impl::allocate_static<_gern_l_x_11, _gern_l_y_12>();

    subtract_vec(max_row_out, _query_input_41, sub_temp);

    auto exp_temp = impl::allocate_static<_gern_l_x_7, _gern_l_y_8>();

    exp_matrix(sub_temp, exp_temp);

    auto sum_row_out = impl::allocate_static_array<_gern_l_x_5>();

    sum_row<stride_val>(sum_row_out, exp_temp);

    divide_vec(sum_row_out, exp_temp, _query_output_40);

    output.insert_new(_gern_x_3, _gern_y_4, _query_output_40);
    // }
}

template<int tile_row,
         int tile_col,
         typename T>
__global__ void softmax_kernel_gern_like(T a,
                                         T b) {
    int x = blockIdx.x;
    int y = threadIdx.y;

    constexpr int64_t num_cols_q = CEILING(a.row, tile_col);
    constexpr int64_t num_rows_in = tile_row;

    auto reg_array_big = a.template query_new<num_rows_in, num_cols_q>(x, y);
    auto output_query = b.template query_new<num_rows_in, num_cols_q>(x, y);

    auto max_row_out = impl::allocate_static_array<num_rows_in>();
    max_shuffle<tile_col>(max_row_out, reg_array_big);

    auto sub_temp = impl::allocate_static<num_rows_in, num_cols_q>();
    subtract_vec(max_row_out, reg_array_big, sub_temp);

    auto exp_temp = impl::allocate_static<num_rows_in, num_cols_q>();
    exp_matrix(sub_temp, exp_temp);

    auto sum_row_out = impl::allocate_static_array<num_rows_in>();
    sum_row<tile_col>(sum_row_out, exp_temp);

    divide_vec(sum_row_out, exp_temp, output_query);

    b.insert_new(x, y, output_query);
}

constexpr int warm_up_runs = 5;
constexpr int kernel_repeats = 5;

#ifndef WIDTH
#define WIDTH 16384
#endif

#ifndef BLOCK_DIM_Y
#define BLOCK_DIM_Y 1024
#endif

#include "original.cuh"

int main() {
    constexpr int64_t h = WIDTH;
    constexpr int64_t w = WIDTH;
    constexpr int tile_col = BLOCK_DIM_Y;

    using MatrixType = impl::MatrixGPU<h, w, h, tile_col>;
    std::cout << WIDTH << std::endl;

    MatrixType in;
    in.ascending();
    MatrixType out;
    out.vvals(0.0f);

    dim3 block_size = dim3(1, BLOCK_DIM_Y, 1);
    dim3 grid_size = dim3(h, 1, 1);

    constexpr int tile_row = 1;

    hipStream_t stream = NULL;

    softmax_kernel10<float><<<grid_size, block_size>>>(in.data, out.data, w, h);
    impl::MatrixCPU reference = out.get();
    out.vvals(0.0f);

    auto specialized = softmax_kernel_gern_like<tile_row, tile_col, MatrixType>;
    // auto specialized = function_39<w, w, 1, tile_col>;
    specialized<<<grid_size, block_size>>>(in, out);
    impl::MatrixCPU gern = out.get();

    for (int64_t i = 0; i < h * w; i++)
        assert(reference.data[i] == gern.data[i]);

    double time = benchmark::measure::execution(
        [&](hipStream_t stream) {
            specialized<<<grid_size, block_size>>>(in, out);
        },
        warm_up_runs,
        kernel_repeats,
        stream);

    double gflops = sizeof(float) * h * w * 2 * 1e-9;
    std::cout << gflops / (time / 1000) << std::endl;

    // specialized = softmax_kernel_mine<tile_row, tile_col, MatrixType>;
    // double time = benchmark::measure::execution(
    //     [&](hipStream_t stream) {
    //         specialized<<<grid_size, block_size>>>(in, out);
    //     },
    //     warm_up_runs,
    //     kernel_repeats,
    //     stream);
    // std::cout << gflops / (time / 1000) << std::endl;

    out.destroy();
    in.destroy();
    reference.destroy();
    gern.destroy();
}