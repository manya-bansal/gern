#include "hip/hip_runtime.h"
// From https:github.com/SzymonOzog/FastSoftmax.git

#include "benchmark.h"
#include "impl/gpu-matrix-const.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<int stride,
         typename T,
         typename T2>
__device__ void query(T a,
                      int x,
                      int y,
                      T2 &reg_array_big) {
    auto &reg_array = reg_array_big.array;
    float4 *val_ptr = reinterpret_cast<float4 *>(&a.data[x * a.row + y * 4]);

    constexpr int64_t num_row = reg_array_big.rows;
    constexpr int64_t num_col = reg_array_big.cols_by_4;

    int index = 0;
    for (int m = 0; m < num_row; m++) {
#pragma unroll URF
        for (int i = 0; i < num_col; i++) {
            float4 val = val_ptr[index];
            reg_array[i] = val;
            index += stride;
        }
    }
}

template<int stride,
         typename T1,
         typename T2>
__device__ void insert(T1 a,
                       int x,
                       int y,
                       T2 &reg_array_big) {
    auto &reg_array = reg_array_big.array;
    constexpr int64_t num_row = reg_array_big.rows;
    constexpr int64_t num_col = reg_array_big.cols_by_4;
    float4 *val_ptr = reinterpret_cast<float4 *>(&a.data[x * a.row + y * 4]);

    int index = 0;
    for (int m = 0; m < num_row; m++) {
#pragma unroll URF
        for (int i = 0; i < num_col; i++) {
            float4 val = reg_array[i];
            val_ptr[index] = val;
            index += stride;
        }
    }
}

template<int num_row, int num_col>
__device__ StaticMatrix<num_row, num_col> allocate_local() {
    return StaticMatrix<num_row, num_col>();
}

#include "impl/impl.h"

template<int tile_row,
         int tile_col,
         typename T>
__global__ void softmax_kernel_mine(T a,
                                    T b) {
    int x = blockIdx.x;
    int y = threadIdx.y;

    constexpr int64_t num_cols_in = CEILING((a.col / 4), tile_col);
    constexpr int64_t num_rows_in = tile_row;

    // if (y < h)
    // {
    StaticMatrix<num_rows_in, num_cols_in> reg_array_big;
    query<tile_col>(a, x, y, reg_array_big);
    holder<num_rows_in> hold;
    max_shuffle<tile_col>(hold, reg_array_big);
    subtract_vec(hold, reg_array_big, reg_array_big);
    exp_matrix(reg_array_big, reg_array_big);
    sum_row<tile_col>(hold, reg_array_big);
    divide_vec(hold, reg_array_big, reg_array_big);
    insert<tile_col>(b, x, y, reg_array_big);
    // }
}

template<int64_t col, int64_t col_val, int64_t row, int64_t stride_val>
__global__ void function_39(impl::MatrixGPU<16384, 16384, 16384, 1024> input, impl::MatrixGPU<16384, 16384, 16384, 1024> output) {

    int64_t _gern_x_3_38_50 = ((blockIdx.x * row) + 0);
    int64_t _gern_y_4_41 = ((threadIdx.y * col_val) + 0);  // MB: CHANGE 1:  threadIdx.x -> threadIdx.y
    int64_t _gern_y_4 = _gern_y_4_41;
    constexpr int64_t _gern_l_y_2 = col_val;
    int64_t _gern_x_3 = _gern_x_3_38_50;
    constexpr int64_t _gern_l_x_1 = row;

    int64_t _gern_x_7_11 = _gern_x_3;
    constexpr int64_t _gern_l_x_5_9 = _gern_l_x_1;
    int64_t _gern_x_16 = _gern_x_3;
    int64_t _gern_y_17 = _gern_y_4;
    constexpr int64_t _gern_l_x_14 = _gern_l_x_1;
    constexpr int64_t _gern_l_y_15 = _gern_l_y_2;
    int64_t _gern_x_20 = _gern_x_16;
    int64_t _gern_y_21 = _gern_y_17;
    constexpr int64_t _gern_l_x_18 = _gern_l_x_14;
    constexpr int64_t _gern_l_y_19 = _gern_l_y_15;
    int64_t _gern_x_24_28 = _gern_x_20;
    constexpr int64_t _gern_l_x_22_26 = _gern_l_x_18;
    auto _query_output_56 = output.template query_new<_gern_l_x_1, _gern_l_y_2>(_gern_x_3, _gern_y_4);

    auto max_row_out = impl::allocate_static_array<_gern_l_x_22_26>();

    int64_t _gern_y_25_29 = ((threadIdx.y * col_val) + 0);
    constexpr int64_t _gern_l_x_22 = max_row_out.size;
    int64_t _gern_x_24 = _gern_x_3;  // Change #2, this is set up as zero for some godforsaken reason.

    int64_t _gern_y_25 = _gern_y_25_29;
    constexpr int64_t _gern_l_y_23 = col_val;

    auto _query_input_57 = input.template query_new<_gern_l_x_22, col>(_gern_x_24, _gern_y_25);

    max_shuffle<stride_val>(max_row_out, _query_input_57);

    auto sub_temp = impl::allocate_static<_gern_l_x_18, _gern_l_y_19>();

    auto _query_input_58 = input.template query_new<_gern_l_x_18, _gern_l_y_19>(_gern_x_20, _gern_y_21);

    subtract_vec(max_row_out, _query_input_58, sub_temp);

    auto exp_temp = impl::allocate_static<_gern_l_x_14, _gern_l_y_15>();

    exp_matrix(sub_temp, exp_temp);

    auto sum_row_out = impl::allocate_static_array<_gern_l_x_5_9>();

    int64_t _gern_y_8_12 = ((threadIdx.y * col_val) + 0);
    constexpr int64_t _gern_l_x_5 = sum_row_out.size;
    int64_t _gern_x_7 = 0;

    int64_t _gern_y_8 = _gern_y_8_12;
    constexpr int64_t _gern_l_y_6 = col_val;

    auto _query_exp_temp_59 = exp_temp.template query_new<_gern_l_x_5, col>(_gern_x_7, _gern_y_8);

    sum_row<stride_val>(sum_row_out, _query_exp_temp_59);

    divide_vec(sum_row_out, exp_temp, _query_output_56);

    output.template insert_new(_gern_x_3, _gern_y_4, _query_output_56);
}

template<int tile_row,
         int tile_col,
         typename T>
__global__ void softmax_kernel_gern_like(T a,
                                         T b) {
    int x = blockIdx.x;
    int y = threadIdx.y;

    constexpr int64_t num_cols_q = CEILING(a.row, tile_col);
    constexpr int64_t num_rows_in = tile_row;

    auto reg_array_big = query<num_rows_in, num_cols_q>(a, x, y);
    auto output_query = query<num_rows_in, num_cols_q>(b, x, y);

    auto max_row_out = impl::allocate_static_array<num_rows_in>();
    max_shuffle<tile_col>(max_row_out, reg_array_big);

    auto sub_temp = impl::allocate_static<num_rows_in, num_cols_q>();
    subtract_vec(max_row_out, reg_array_big, sub_temp);

    auto exp_temp = impl::allocate_static<num_rows_in, num_cols_q>();
    exp_matrix(sub_temp, exp_temp);

    auto sum_row_out = impl::allocate_static_array<num_rows_in>();
    sum_row<tile_col>(sum_row_out, exp_temp);

    divide_vec(sum_row_out, exp_temp, output_query);

    insert(b, x, y, output_query);
}

constexpr int warm_up_runs = 5;
constexpr int kernel_repeats = 5;

#ifndef WIDTH
#define WIDTH 16384
#endif

#ifndef BLOCK_DIM_Y
#define BLOCK_DIM_Y 1024
#endif

#include "original.cuh"

int main() {
    constexpr int64_t h = WIDTH;
    constexpr int64_t w = WIDTH;
    constexpr int tile_col = BLOCK_DIM_Y;

    using MatrixType = impl::MatrixGPU<h, w, h, tile_col>;
    std::cout << WIDTH << std::endl;

    MatrixType in;
    in.ascending();
    MatrixType out;
    out.vvals(0.0f);

    dim3 block_size = dim3(1, BLOCK_DIM_Y, 1);
    dim3 grid_size = dim3(h, 1, 1);

    constexpr int tile_row = 1;

    hipStream_t stream = NULL;

    softmax_kernel10<float><<<grid_size, block_size>>>(in.data, out.data, w, h);
    impl::MatrixCPU reference = out.get();
    out.vvals(0.0f);

    // auto specialized = softmax_kernel_gern_like<tile_row, tile_col, MatrixType>;
    auto specialized = function_39<w / tile_col, w / tile_col, 1, tile_col>;
    specialized<<<grid_size, block_size>>>(in, out);
    impl::MatrixCPU gern = out.get();

    for (int64_t i = 0; i < h * w; i++)
        assert(reference.data[i] == gern.data[i]);

    double time = benchmark::measure::execution(
        [&](hipStream_t stream) {
            specialized<<<grid_size, block_size>>>(in, out);
        },
        warm_up_runs,
        kernel_repeats,
        stream);

    double gflops = sizeof(float) * h * w * 2 * 1e-9;
    std::cout << gflops / (time / 1000) << std::endl;

    // specialized = softmax_kernel_mine<tile_row, tile_col, MatrixType>;
    // double time = benchmark::measure::execution(
    //     [&](hipStream_t stream) {
    //         specialized<<<grid_size, block_size>>>(in, out);
    //     },
    //     warm_up_runs,
    //     kernel_repeats,
    //     stream);
    // std::cout << gflops / (time / 1000) << std::endl;

    out.destroy();
    in.destroy();
    reference.destroy();
    gern.destroy();
}