
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iomanip>

#include "benchmark.h"
#include "sgemm_device.cuh"
#include "shims.cuh"

__host__ void querySharedMemory(int deviceId = 0) {
    hipDeviceProp_t deviceProp;
    hipError_t error = hipGetDeviceProperties(&deviceProp, deviceId);

    if (error != hipSuccess) {
        printf("Error getting device properties: %s\n", hipGetErrorString(error));
        return;
    }

    // Get shared memory per block in bytes
    size_t sharedMemPerBlock = deviceProp.sharedMemPerBlock;

    // Get maximum shared memory per block in bytes (for newer architectures)
    size_t sharedMemPerBlockOptin = deviceProp.sharedMemPerBlockOptin;

    printf("Device %d: %s\n", deviceId, deviceProp.name);
    printf("Default Shared Memory per Block: %zu bytes (%.2f KB)\n",
           sharedMemPerBlock, sharedMemPerBlock / 1024.0);
    printf("Maximum Shared Memory per Block: %zu bytes (%.2f KB)\n",
           sharedMemPerBlockOptin, sharedMemPerBlockOptin / 1024.0);

    // Get current shared memory configuration
    int currentSharedMem;
    hipDeviceGetAttribute(&currentSharedMem,
                           hipDeviceAttributeSharedMemPerBlockOptin,
                           deviceId);
    printf("Current Shared Memory Configuration: %zu bytes (%.2f KB)\n",
           currentSharedMem, currentSharedMem / 1024.0);
}

int main(int argc, char **argv) {
    querySharedMemory();
    constexpr int M = 1024;
    constexpr int N = 1024;
    constexpr int K = 1024;
    constexpr int row_major = true;
    float alpha = 0.5f;
    float beta = 3.0f;

    using MatrixTypeA = impl::MatrixGPU<M, K, K, !row_major>;
    MatrixTypeA a;
    a.ascending();
    using MatrixTypeB = impl::MatrixGPU<K, N, N, row_major>;
    MatrixTypeB b;
    b.ascending();
    using MatrixTypeC = impl::MatrixGPU<M, N, N, row_major>;
    MatrixTypeC c;
    c.vvals(0.0f);

    runSgemmWarptiling(M, N, K, alpha, a.data, b.data, beta, c.data);
    auto ref_c = c.get();

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    CUBLAS_CHECK_AND_EXIT(hipblasCreate(&cublasH));

    CUDA_CHECK_AND_EXIT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK_AND_EXIT(hipblasSetStream(cublasH, stream));

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    c.vvals(0.0f);
    CUBLAS_CHECK_AND_EXIT(hipblasSgemm(cublasH, transa, transb, M, N, K,
                                      &alpha, a.data, K, b.data, N,
                                      &beta, c.data, N));

    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));
    auto ref_blas = c.get();

    CUDA_CHECK_AND_EXIT(hipGetLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    c.vvals(0.0f);
    runSgemmGern(a, b, c, alpha, beta);
    auto ref_gern = c.get();

    for (int i = 0; i < M * N; i++) {
        // std::cout << i << std::endl;
        // std::cout << ref_c.data[i] << std::endl;
        // std::cout << ref_blas.data[i] << std::endl;
        assert(ref_c.data[i] - ref_blas.data[i] < 0.00001f);
        assert(ref_gern.data[i] - ref_blas.data[i] < 0.00001f);
    }
}