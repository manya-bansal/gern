#include <hip/hip_runtime.h>
#include <iostream>

// Kernel that attempts to use an extremely large shared memory allocation
__global__ void excessSharedMemKernel(float *d_out, int size) {
    extern __shared__ float shared_mem[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        shared_mem[tid % blockDim.x] = 1.0f;  // Simple write
        d_out[tid] = shared_mem[tid % blockDim.x];
    }
}

int main() {
    int N = 1024;
    size_t bytes = N * sizeof(float);

    // Allocate host memory
    float *h_out = new float[N];

    // Allocate device memory
    float *d_out;
    hipMalloc(&d_out, bytes);

    // Check GPU shared memory limits
    int sharedMemPerBlockDefault, sharedMemMaxPossible;
    hipDeviceGetAttribute(&sharedMemPerBlockDefault, hipDeviceAttributeSharedMemPerBlockOptin, 0);
    hipDeviceGetAttribute(&sharedMemMaxPossible, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, 0);

    std::cout << "Default max shared memory per block: " << sharedMemPerBlockDefault << " bytes\n";
    std::cout << "Absolute max shared memory per multiprocessor: " << sharedMemMaxPossible << " bytes\n";

    int threadsPerBlock = 256;
    int excessiveSharedMemSize = sharedMemPerBlockDefault + 1;  // Try exceeding the default limit

    // Attempt to set a higher shared memory limit
    hipError_t err = hipFuncSetAttribute(reinterpret_cast<const void*>(excessSharedMemKernel), hipFuncAttributeMaxDynamicSharedMemorySize, excessiveSharedMemSize);
    if (err != hipSuccess) {
        std::cout << "Failed to increase shared memory limit: " << hipGetErrorString(err) << "\n";
    }

    // Launch kernel
    excessSharedMemKernel<<<1, threadsPerBlock, excessiveSharedMemSize>>>(d_out, N);

    // Check for errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error after kernel launch: " << hipGetErrorString(err) << "\n";
    }

    // Cleanup
    hipFree(d_out);
    delete[] h_out;

    return 0;
}
