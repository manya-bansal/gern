#include "hip/hip_runtime.h"
#include "sh_malloc.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>

#ifndef CUDA_CHECK_AND_EXIT
#define CUDA_CHECK_AND_EXIT(error)                                                                      \
    {                                                                                                   \
        auto status = static_cast<hipError_t>(error);                                                  \
        if (status != hipSuccess) {                                                                    \
            std::cout << hipGetErrorString(status) << " " << __FILE__ << ":" << __LINE__ << std::endl; \
            std::exit(status);                                                                          \
        }                                                                                               \
    }
#endif

__global__ void test() {
    extern __shared__ char shmem[];

    init_shmem((size_t)208);

    void *ptr1 = sh_malloc(90);
    // printf("ptr1: %p\n", ptr1);
    // printf("shmem: %p\n", shmem);

    assert(ptr1 == shmem + (sizeof(shmem_meta_data)) + sizeof(size_t));

    for (int i = 0; i < 100; i++) {
        void *ptr = sh_malloc(40);
        // printf("ptr: %p\n", ptr);
        // assert((char *)ptr - (char *)ptr1 == 100 + 4);
        sh_free(ptr);
        // ptr1 = ptr;
    }

    // Should fail
    void *ptr = sh_malloc(100);
}

int main() {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Max shared memory per block: %d bytes\n", prop.sharedMemPerBlockOptin);
    size_t maxSharedMemory = prop.sharedMemPerBlockOptin;

    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(test), hipFuncAttributeMaxDynamicSharedMemorySize, maxSharedMemory));

    dim3 block(1);
    dim3 grid(1);

    test<<<grid, block, maxSharedMemory>>>();
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " " << __FILE__ << ":" << __LINE__ << std::endl;
        std::exit(err);
    }
}